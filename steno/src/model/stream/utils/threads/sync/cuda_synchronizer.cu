#include "cuda_synchronizer.h"

namespace Model
{
CudaSynchronizer::CudaSynchronizer(hipStream_t stream)
    : stream_(stream)
{
}

void CudaSynchronizer::sync() const
{
    hipStreamSynchronize(stream_);
}

}    // namespace Model