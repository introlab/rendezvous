#include "cuda_image_file_reader.h"

namespace Model
{
CudaImageFileReader::CudaImageFileReader(const std::string& imageFilePath, ImageFormat format)
    : ImageFileReader(imageFilePath, format)
{
    deviceCudaObjectFactory_.allocateObject(image_);
    hipMemcpy(image_.deviceData, image_.hostData, image_.size, hipMemcpyHostToDevice);
}

CudaImageFileReader::~CudaImageFileReader()
{
    deviceCudaObjectFactory_.deallocateObject(image_);
}

const Image& CudaImageFileReader::readImage()
{
    return ImageFileReader::readImage();
}
}    // namespace Model