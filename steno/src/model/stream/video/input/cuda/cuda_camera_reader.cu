#include "cuda_camera_reader.h"

#include "model/stream/utils/cuda_utils.cuh"

namespace Model
{
CudaCameraReader::CudaCameraReader(const VideoConfig& videoConfig)
    : CameraReader(videoConfig, 3)
    , nextImage_(nullptr)
{
    for (std::size_t i = 0; i < images_.size(); ++i)
    {
        deviceCudaObjectFactory_.allocateObject(images_.current().image);
        images_.next();
    }

    checkCuda(hipStreamCreate(&stream_));
}

CudaCameraReader::~CudaCameraReader()
{
    for (std::size_t i = 0; i < images_.size(); ++i)
    {
        deviceCudaObjectFactory_.deallocateObject(images_.current().image);
        images_.next();
    }

    hipStreamDestroy(stream_);
}

void CudaCameraReader::open()
{
    CameraReader::open();

    // Read a frame to prepare the next read
    nextImage_ = &CameraReader::readImage();
    hipMemcpyAsync(nextImage_->deviceData, nextImage_->hostData, nextImage_->size, hipMemcpyHostToDevice, stream_);
}

const Image& CudaCameraReader::readImage()
{
    if (nextImage_ == nullptr)
    {
        throw std::runtime_error("Camera reader frame is null, this is not supposed to occur!");
    }

    const Image& image = *nextImage_;
    hipStreamSynchronize(stream_);

    nextImage_ = &CameraReader::readImage();
    hipMemcpyAsync(nextImage_->deviceData, nextImage_->hostData, nextImage_->size, hipMemcpyHostToDevice, stream_);

    return image;
}
}    // namespace Model