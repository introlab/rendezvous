#include "cuda_camera_reader.h"

#include "model/stream/utils/cuda_utils.cuh"

#include <cstring>

namespace Model
{
CudaCameraReader::CudaCameraReader(std::shared_ptr<VideoConfig> videoConfig)
    : CameraReader(videoConfig, 3)
    , nextImage_(nullptr)
    , pageLockedImage_(videoConfig->resolution.width, videoConfig->resolution.height, videoConfig->imageFormat)
{
    checkCuda(hipHostMalloc(&pageLockedImage_.hostData, pageLockedImage_.size, 0));

    for (std::size_t i = 0; i < images_.size(); ++i)
    {
        deviceCudaObjectFactory_.allocateObject(images_.current().image);
        images_.next();
    }

    checkCuda(hipStreamCreate(&stream_));
}

CudaCameraReader::~CudaCameraReader()
{
    hipHostFree(pageLockedImage_.hostData);

    for (std::size_t i = 0; i < images_.size(); ++i)
    {
        deviceCudaObjectFactory_.deallocateObject(images_.current().image);
        images_.next();
    }

    hipStreamDestroy(stream_);
}

void CudaCameraReader::open()
{
    CameraReader::open();
    nextImage_ = &CameraReader::readImage();
    copyImageToDevice(*nextImage_);
    hipStreamSynchronize(stream_);
}

void CudaCameraReader::close()
{
    CameraReader::close();
    nextImage_ = nullptr;
}

const Image& CudaCameraReader::readImage()
{
    if (nextImage_ == nullptr)
    {
        throw std::runtime_error("Camera reader frame is null, this is not supposed to occur!");
    }

    const Image& image = *nextImage_;
    nextImage_ = &CameraReader::readImage();
    copyImageToDevice(*nextImage_);

    return image;
}

void CudaCameraReader::copyImageToDevice(const Image& image)
{
    // Copy the image data to a page-locked image (this is for faster async copy to device memory)
    std::memcpy(pageLockedImage_.hostData, image.hostData, image.size);

    // Wait for the previous async copy completion
    hipStreamSynchronize(stream_);

    // Async copy to device memory
    hipMemcpyAsync(image.deviceData, pageLockedImage_.hostData, image.size, hipMemcpyHostToDevice, stream_);
}
}    // namespace Model