#include "vc_cuda_camera_reader.h"

#include "model/stream/utils/cuda_utils.cuh"

#include <cstring>

namespace Model
{
VcCudaCameraReader::VcCudaCameraReader(std::shared_ptr<VideoConfig> videoConfig)
    : VcCameraReader(videoConfig, 3)
    , pageLockedImage_(videoConfig->resolution.width, videoConfig->resolution.height, videoConfig->imageFormat)
{
    checkCuda(hipHostMalloc(&pageLockedImage_.hostData, pageLockedImage_.size, 0));

    for (std::size_t i = 0; i < images_.size(); ++i)
    {
        deviceCudaObjectFactory_.allocateObject(images_.current());
        images_.next();
    }

    checkCuda(hipStreamCreate(&stream_));
}

VcCudaCameraReader::~VcCudaCameraReader()
{
    hipHostFree(pageLockedImage_.hostData);

    for (std::size_t i = 0; i < images_.size(); ++i)
    {
        deviceCudaObjectFactory_.deallocateObject(images_.current());
        images_.next();
    }

    hipStreamDestroy(stream_);
}

void VcCudaCameraReader::open()
{
    VcCameraReader::open();
    VcCameraReader::readImage(nextImage_);
    copyImageToDevice(nextImage_);
    hipStreamSynchronize(stream_);
}

void VcCudaCameraReader::close()
{
    BaseCameraReader::close();
}

bool VcCudaCameraReader::readImage(Image& image)
{
    image = nextImage_;
    VcCameraReader::readImage(nextImage_);
    copyImageToDevice(nextImage_);

    return true;
}

void VcCudaCameraReader::copyImageToDevice(const Image& image)
{
    // Copy the image data to a page-locked image (this is for faster async copy to device memory)
    std::memcpy(pageLockedImage_.hostData, image.hostData, image.size);

    // Wait for the previous async copy completion
    hipStreamSynchronize(stream_);

    // Async copy to device memory
    hipMemcpyAsync(image.deviceData, pageLockedImage_.hostData, image.size, hipMemcpyHostToDevice, stream_);
}
}    // namespace Model