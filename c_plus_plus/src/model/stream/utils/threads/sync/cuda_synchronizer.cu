#include "cuda_synchronizer.h"

CudaSynchronizer::CudaSynchronizer(hipStream_t stream)
    : stream_(stream)
{
}

void CudaSynchronizer::sync() const { hipStreamSynchronize(stream_); }