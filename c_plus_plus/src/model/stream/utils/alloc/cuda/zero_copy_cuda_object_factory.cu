#include "zero_copy_cuda_object_factory.h"

#include "model/stream/utils/cuda_utils.cuh"

namespace
{
template <typename T>
void mallocHost(T*& hostPtr, std::size_t size)
{
    checkCuda(hipHostAlloc(&hostPtr, size * sizeof(T), hipHostMallocMapped));
}

template <typename T>
void deallocHost(T*& hostPtr)
{
    checkCuda(hipHostFree(hostPtr));
    hostPtr = nullptr;
}

template <typename T>
void getDevicePointer(T* hostPtr, T*& devicePtr)
{
    checkCuda(hipHostGetDevicePointer(&devicePtr, hostPtr, 0));
}
}    // namespace

ZeroCopyCudaObjectFactory::ZeroCopyCudaObjectFactory() { hipSetDeviceFlags(hipDeviceMapHost); }

void ZeroCopyCudaObjectFactory::allocateObject(Image& image) const
{
    mallocHost(image.hostData, image.size);
    getDevicePointer(image.hostData, image.deviceData);
}

void ZeroCopyCudaObjectFactory::deallocateObject(Image& image) const
{
    deallocHost(image.hostData);
    image.deviceData = nullptr;
}

void ZeroCopyCudaObjectFactory::allocateObject(ImageFloat& image) const
{
    mallocHost(image.hostData, image.size);
    getDevicePointer(image.hostData, image.deviceData);
}

void ZeroCopyCudaObjectFactory::deallocateObject(ImageFloat& image) const
{
    deallocHost(image.hostData);
    image.deviceData = nullptr;
}

void ZeroCopyCudaObjectFactory::allocateObject(DewarpingMapping& mapping) const
{
    mallocHost(mapping.hostData, mapping.size);
    getDevicePointer(mapping.hostData, mapping.deviceData);
}

void ZeroCopyCudaObjectFactory::deallocateObject(DewarpingMapping& mapping) const
{
    deallocHost(mapping.hostData);
    mapping.deviceData = nullptr;
}

void ZeroCopyCudaObjectFactory::allocateObject(FilteredDewarpingMapping& mapping) const
{
    mallocHost(mapping.hostData, mapping.size);
    getDevicePointer(mapping.hostData, mapping.deviceData);
}

void ZeroCopyCudaObjectFactory::deallocateObject(FilteredDewarpingMapping& mapping) const
{
    deallocHost(mapping.hostData);
    mapping.deviceData = nullptr;
}