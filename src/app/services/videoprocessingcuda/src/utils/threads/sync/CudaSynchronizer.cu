#include "CudaSynchronizer.h"

CudaSynchronizer::CudaSynchronizer(hipStream_t stream)
    : stream_(stream)
{
}

void CudaSynchronizer::sync() const
{
    hipStreamSynchronize(stream_);
}